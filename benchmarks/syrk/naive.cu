#include "hip/hip_runtime.h"

#include "../../timing/dphpc_timing.h"

#define get(A, ncols, r, c) A[(r)*(ncols)+(c)]

__global__ void syrk(int n, int k, double alpha, double beta, double *C, double *A) {
    int c = blockIdx.x * blockDim.x + threadIdx.x;
    int r = blockIdx.y * blockDim.y + threadIdx.y;
    if (r < n && c < n && r >= c) {
        double s = 0.0;
        for (int i = 0; i < k; i++) {
            s += get(A, k, r, i) * get(A, k, c, i);
        }
        get(C, n, r, c) = beta * get(C, n, r, c) + alpha * s;
    }
}

void init_array(int n, int m, double *alpha, double *beta,
    double* C, double* A)
{
    *alpha = 1.5;
    *beta = 1.2;
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < m; j++) {
            get(A, m, i, j) = (double) ((i*j+1)%n) / n;
        }
    }
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++) {
            get(C, n, i, j) = (double) ((i*j+2)%m) / m;
        }
    }
}

void run_kernel(int n, int k, double alpha, double beta, double *C, double *A) {
    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((n-1) / 16 + 1, (n-1) / 16 + 1);
    syrk<<<numBlocks,threadsPerBlock>>>(n, k, alpha, beta, C, A); 
    hipDeviceSynchronize();
}


void run_bm(int n, int m, const char* preset) {
    double alpha;
    double beta;
    
    double *C = (double*) malloc(n*n*sizeof(double));
    double *A = (double*) malloc(n*m*sizeof(double));

    double *C_d, *A_d;
    hipMalloc((void**) &C_d, n*n*sizeof(double));
    hipMalloc((void**) &A_d, n*m*sizeof(double));

    init_array(n, m, &alpha, &beta, C, A);

    hipMemcpy((void*) C_d, (void*) C, n*n*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy((void*) A_d, (void*) A, n*m*sizeof(double), hipMemcpyHostToDevice);

    dphpc_time3(
        hipMemcpy(C_d, C, n*n*sizeof(double), hipMemcpyHostToDevice);,
        run_kernel(n, m, alpha, beta, C_d, A_d),
        preset
    );

    hipFree((void*) C_d);
    hipFree((void*) A_d);

    free((void*)C);
    free((void*)A);
}


int is_valid() {

    int n = 200;
    int m = 70;

    double alpha = 3.0;
    double beta = 5.0;
    
    double *C = (double*) malloc(n*n*sizeof(double));
    double *A = (double*) malloc(n*m*sizeof(double));

    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++) {
            get(C, n, i, j) = 1;
        }
        for (int j = 0; j < m; j++) {
            get(A, m, i, j) = 1;
        }
    } 

    double *C_d, *A_d;
    hipMalloc((void**) &C_d, n*n*sizeof(double));
    hipMalloc((void**) &A_d, n*m*sizeof(double));

    hipMemcpy((void*) C_d, (void*) C, n*n*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy((void*) A_d, (void*) A, n*m*sizeof(double), hipMemcpyHostToDevice);

    run_kernel(n, m, alpha, beta, C_d, A_d);

    hipMemcpy((void*) C, (void*) C_d, n*n*sizeof(double), hipMemcpyDeviceToHost);

    hipDeviceSynchronize();

    hipFree((void*) C_d);
    hipFree((void*) A_d);

    free((void*)A);

    

    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++) {
            if (j <= i && get(C, n, i, j) != beta + alpha * m) {
                free((void*)C);
                printf("validation failed");
                return 0;
            }
        }
    } 

    free((void*)C);
    return 1;
}

#include "_main.h"

