#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#include "../../timing/dphpc_timing.h"

#define DEV_MODE 0
#define TIME 1

// CUDA kernel to calculate covariance matrix
__global__ void covariance_kernel(int M, int N, double* data, double* cov) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < M && j < M) {
        double partial_sum = 0.0;

        for (int k = 0; k < N; k++) {
            partial_sum += data[k * M + i] * data[k * M + j];
        }

        cov[i * M + j] = partial_sum / ((double)N - 1.0);
    }
}

__global__ void mean_adjust_kernel(double *data, int N, int M) {
    int col_idx = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (int j = col_idx; j < M; j += stride) {
        double local_sum = 0.0;

        for (int row_idx = 0; row_idx < N; ++row_idx) {
            local_sum += data[row_idx * M + j];
        }

        double mean_j = local_sum / N;

        for (int row_idx = 0; row_idx < N; ++row_idx) {
            data[row_idx * M + j] -= mean_j;
        }
    }
}


void reset(int M, int N, double* data, double* d_data) {
    hipMemcpy(d_data, data, N * M * sizeof(double), hipMemcpyHostToDevice);
    hipDeviceSynchronize();
}

void initialize(int M, int N, double* data) {
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < M; j++) {
            data[i * M + j] = (double)(i * j) / M;
        }
    }
}

void printMatrix(int n, int m, double *matrix) {
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < m; j++) {
            printf("%.6lf ", matrix[i * m + j]);
        }
        printf("\n");
    }
}

void kernel(int M, int N, double* d_data, double* d_cov) {
    int threads_per_block = 256;
    int blocks = (M + threads_per_block - 1) / threads_per_block;

    mean_adjust_kernel<<<blocks, threads_per_block>>>(d_data, N, M);
    hipDeviceSynchronize();


    dim3 blockDim(16, 16);
    dim3 gridDim((M + blockDim.x - 1) / blockDim.x, (M + blockDim.y - 1) / blockDim.y);

    covariance_kernel<<<gridDim, blockDim>>>(M, N, d_data, d_cov);
    hipDeviceSynchronize();

    // hipMemcpy(cov, d_cov, M * M * sizeof(double), hipMemcpyDeviceToHost);
    // hipFree(d_data);
    // hipFree(d_cov);
    // free(mean);
}

void run_bm(int M, int N, const char* preset) {
    double *data = (double*)malloc(N * M * sizeof(double));
    double *cov = (double*)malloc(M * M * sizeof(double));
    initialize(M, N, data);

    double *d_data, *d_cov;
    hipMalloc((void**)&d_cov, M * M * sizeof(double));
    hipMalloc((void**)&d_data, N * M * sizeof(double));


    #if DEV_MODE
    printf("Data matrix: \n");
    printMatrix(N, M, data);
    #endif

    #if TIME
        dphpc_time3(
            reset(M, N, data, d_data),
            kernel(M, N, d_data, d_cov),
            preset
        );
    #else
        reset(M, N, data, d_data);
        kernel(M, N, d_data, d_cov);
    #endif

    hipMemcpy(cov, d_cov, M * M * sizeof(double), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    #if DEV_MODE
    printf("Covariance matrix: \n");
    printMatrix(M, M, cov);
    printf("END\n\n");
    #endif

    hipFree(d_data);
    hipFree(d_cov);
    free(data);
    free(cov);
}

int main() {
    #if DEV_MODE
        run_bm(3, 4, "M");
        run_bm(5, 7, "M");
        // run_bm(3, 4, "S");
        // run_bm(5, 5, "S");
        // run_bm(5, 7, "M");
    #else
        run_bm(500, 600, "S");
        run_bm(1400, 1800, "M");
        run_bm(3200, 4000, "L");
        run_bm(1200, 1400, "paper");
    #endif

    return 0;
}
