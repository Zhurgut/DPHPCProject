#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

// #include "utils.h"
#include "../../timing/dphpc_timing.h"

__global__ void scalar_update_kernel(double *x, int i, double *b, double *dp, double *L, int N) {
    x[i] = (b[i] - dp[0]) / L[i * N + i];
}

__global__ void dot_product_kernel(double *L, double *x, double *dp, int i, int N) {
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    if (j < i) {
        dp[j] = L[i * N + j] * x[j]; 
        __syncthreads();

        for (int stride = 1; j + stride < i; stride <<= 1) {
            if (j % (2 * stride) == 0){
                dp[j] += dp[j + stride]; 
            }
            __syncthreads();
        }
    }
}

void kernel(double *L, double *x, double *b, int N) {
    int blockSize = 256;
    int numBlocks = (N + blockSize - 1) / blockSize;

    double *dp;
    hipMalloc((void **)&dp, N * sizeof(double));

    for(int i = 0; i < N; i++) {
        dot_product_kernel<<<numBlocks, blockSize>>>(L, x, dp, i, N);
        scalar_update_kernel<<<1, 1>>>(x, i, b, dp, L, N);
    }

    hipDeviceSynchronize();
}

void reset(double *L, double *x, double *b, double *d_L, double *d_x, double *d_b, int N) {
    hipMemcpy(d_L, L, N * N * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_x, x, N * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, N * sizeof(double), hipMemcpyHostToDevice);
    hipDeviceSynchronize();
}

void initialize(int N, double *L, double *x, double *b) {
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            L[i * N + j] = (i + N - j + 1) * 2.0 / N;
        }
        x[i] = 0;
        b[i] = (double)i;
    }
}

void run_bm(int N, const char *preset) {
    double *L = (double*)malloc(sizeof(double) * N * N);
    double *x = (double*)malloc(sizeof(double) * N);
    double *b = (double*)malloc(sizeof(double) * N);

    initialize(N, L, x, b);

    double *d_L, *d_x, *d_b;
    hipMalloc((void **)&d_L, N * N * sizeof(double));
    hipMalloc((void **)&d_x, N * sizeof(double));
    hipMalloc((void **)&d_b, N * sizeof(double));

    reset(L, x, b, d_L, d_x, d_b, N);
    kernel(d_L, d_x, d_b, N);
    hipDeviceSynchronize();


    dphpc_time3(
        reset(L, x, b, d_L, d_x, d_b, N),
        kernel(d_L, d_x, d_b, N),
        preset
    );


    hipMemcpy(x, d_x, N * sizeof(double), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    
    printf("Done \n");
    for (int i = 0; i < 5; i++) {
        printf("%f ", x[i]);
    }
    
    printf("%f ", x[N-1]);

    hipFree(d_L);
    hipFree(d_x);
    hipFree(d_b);
    free(L);
    free(x);
    free(b);
}

int main() {
    run_bm(2000, "S");
    run_bm(5000, "M");
    run_bm(14000, "L");
    run_bm(16000, "paper");

    return 0;
}


