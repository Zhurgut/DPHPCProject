#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#include "utils.h"
#include "../../timing/dphpc_timing.h"

bool VALIDATE = false;

__global__ void scalar_update_kernel(double *L, double *x, double *b, double *dp, int N, int i) {
    x[i] = (b[i] - dp[0]) / L[i * N + i];
}

__global__ void dot_product_kernel(double *L, double *x, double *dp, int i, int N) {
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    if (j < i) {
        dp[j] = L[i * N + j] * x[j]; 
        __syncthreads();

        for (int stride = 1; j + stride < i; stride <<= 1) {
            if (j % (2 * stride) == 0){
                dp[j] += dp[j + stride]; 
            }
            __syncthreads();
        }
    }
}

void kernel(double *L, double *x, double *b, int N) {
    int blockSize = 256;
    int numBlocks = (N + blockSize - 1) / blockSize;

    double *dp;
    hipMalloc((void **)&dp, N * sizeof(double));

    for(int i = 0; i < N; i++) {
        dot_product_kernel<<<numBlocks, blockSize>>>(L, x, dp, i, N);
        hipDeviceSynchronize();
        scalar_update_kernel<<<1, 1>>>(L, x, b, dp, N, i);
        hipDeviceSynchronize();
    }

    hipDeviceSynchronize();
    hipFree(dp);
}

void reset(double *L, double *x, double *b, double *d_L, double *d_x, double *d_b, int N) {
    initialize(N, L, x, b);

    hipMemcpy(d_L, L, N * N * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_x, x, N * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, N * sizeof(double), hipMemcpyHostToDevice);
    hipDeviceSynchronize();
}

void run_bm(int N, const char *preset) {
    double *L = (double*)malloc(sizeof(double) * N * N);
    double *x = (double*)malloc(sizeof(double) * N);
    double *b = (double*)malloc(sizeof(double) * N);

    double *d_L, *d_x, *d_b;
    hipMalloc((void **)&d_L, N * N * sizeof(double));
    hipMalloc((void **)&d_x, N * sizeof(double));
    hipMalloc((void **)&d_b, N * sizeof(double));

    if (VALIDATE) {
        reset(L, x, b, d_L, d_x, d_b, N);

        hipMemcpy(d_x, x, N * sizeof(double), hipMemcpyHostToDevice);
        hipDeviceSynchronize();
        kernel(d_L, d_x, d_b, N);

        hipMemcpy(x, d_x, N * sizeof(double), hipMemcpyDeviceToHost);
        hipDeviceSynchronize();

        hipError_t cudaErr = hipGetLastError();
        if (cudaErr != hipSuccess) {
            fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(cudaErr));
        }

        if (!is_correct(N, x, preset)) {
            printf("Validation failed for preset: %s \n", preset);
            exit(1);
        } else {
            printf("Validation passed for preset: %s \n", preset);
        }
    }

    dphpc_time3(
        reset(L, x, b, d_L, d_x, d_b, N),
        kernel(d_L, d_x, d_b, N),
        preset
    );

    hipFree(d_L);
    hipFree(d_x);
    hipFree(d_b);
    free(L);
    free(x);
    free(b);
}

int main() {
    const char *presets[] = {"S", "M", "L", "paper"};

    for (int i = 0; i < 4; i++) {
        const char* preset = presets[i];
        int n = get_params(preset)[0];
        run_bm(n, preset);
    }

    return 0;
}


