#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#include "utils.h"
#include "../../timing/dphpc_timing.h"

bool VALIDATE = false;

__global__ void kernel(double *L, double *x, double *b, int i, int N) {
    int id = blockIdx.x * blockDim.x + threadIdx.x + 1;
    int j  = i + id - 1;
    int d = i-1;

    if (d >= 1 && j <= N) {
        x[j-1] += x[d-1]*L[(j-1)*N + d-1];
    }

    if (id == 1) {
        x[i-1] = (b[i-1] - x[i-1]) / L[(i-1)*N + i-1];
    }
}

void kernel(double *L, double *x, double *b, int N) {
    int threads = 64;
    

    for(int i = 1; i <= N; i++) {
        int numBlocks = (N - i) / threads + 1;
        kernel<<<numBlocks, threads>>>(L, x, b, N, i);
    }

    hipDeviceSynchronize();
}

void reset(double *L, double *x, double *b, double *d_L, double *d_x, double *d_b, int N) {
    initialize(N, L, x, b);

    hipMemcpy(d_L, L, N * N * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_x, x, N * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, N * sizeof(double), hipMemcpyHostToDevice);
    hipDeviceSynchronize();
}

void run_bm(int N, const char *preset) {
    double *L = (double*)malloc(sizeof(double) * N * N);
    double *x = (double*)malloc(sizeof(double) * N);
    double *b = (double*)malloc(sizeof(double) * N);

    double *d_L, *d_x, *d_b;
    hipMalloc((void **)&d_L, N * N * sizeof(double));
    hipMalloc((void **)&d_x, N * sizeof(double));
    hipMalloc((void **)&d_b, N * sizeof(double));

    if (VALIDATE) {
        reset(L, x, b, d_L, d_x, d_b, N);

        hipMemcpy(d_x, x, N * sizeof(double), hipMemcpyHostToDevice);
        hipDeviceSynchronize();
        kernel(d_L, d_x, d_b, N);

        hipMemcpy(x, d_x, N * sizeof(double), hipMemcpyDeviceToHost);
        hipDeviceSynchronize();

        hipError_t cudaErr = hipGetLastError();
        if (cudaErr != hipSuccess) {
            fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(cudaErr));
        }

        if (!is_correct(N, x, preset)) {
            printf("Validation failed for preset: %s \n", preset);
            exit(1);
        } else {
            printf("Validation passed for preset: %s \n", preset);
        }
    }

    dphpc_time3(
        reset(L, x, b, d_L, d_x, d_b, N),
        kernel(d_L, d_x, d_b, N),
        preset
    );

    hipFree(d_L);
    hipFree(d_x);
    hipFree(d_b);
    free(L);
    free(x);
    free(b);
}

int main() {
    const char *presets[] = {"S", "M", "L", "paper"};

    for (int i = 0; i < 4; i++) {
        const char* preset = presets[i];
        int n = get_params(preset)[0];
        run_bm(n, preset);
    }

    return 0;
}


