#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#include "utils.h"
#include "../../timing/dphpc_timing.h"

bool VALIDATE = false;

__global__ void inv_diag_kernel(double *matrix, double *diag, int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N) {
        diag[i] = 1.0 / matrix[i * N + i];
    }
}

__global__ void scalar_mult_kernel(double *L, double *Lx, double *scalar, int j, int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N) {
        Lx[i * N + j] = L[i * N + j] * scalar[0];
    }
}

__global__ void pre_comp_kernel(double *L, double *Lx, double *x, int j, int N) {
    int start_row = j;
    int i = blockIdx.x * blockDim.x + threadIdx.x + start_row;
    if (i < N) {
        Lx[i * N + j] = L[i * N + j] * x[j] + Lx[i * N + (j - 1)];
    }
}

__global__ void scalar_update_kernel(int N, double *x, int i, double *Lx, double *inv_diag) {
    x[i] -= (Lx[i * N + (i - 1)] * inv_diag[i]);
}

__global__ void element_wise_mult_kernel(double *a, double *b, double *dst, int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x; 
    if (i < N) {
        dst[i] = a[i] * b[i];
    }
}

void kernel(double *d_L, double *d_x, double *d_b, int N) {
    int t = 256;
    int blocks = (N + t - 1) / t;

    double *d_inv_diag, *d_b_prod_inv_diag, *d_Lx;
    hipMalloc((void **)&d_inv_diag, N * sizeof(double));
    hipMalloc((void **)&d_b_prod_inv_diag, N * sizeof(double));
    hipMalloc((void **)&d_Lx, N * N * sizeof(double));

    inv_diag_kernel<<<blocks, t>>>(d_L, d_inv_diag, N);
    hipDeviceSynchronize();

    element_wise_mult_kernel<<<blocks, t>>>(d_b, d_inv_diag, d_b_prod_inv_diag, N);
    hipDeviceSynchronize();

    // d_x = d_b_prod_inv_diag; // now wrong

    scalar_mult_kernel<<<blocks, t>>>(d_L, d_Lx, d_b_prod_inv_diag, 0, N);
    hipDeviceSynchronize();

    for (int i = 1; i < N; i++) {
        scalar_update_kernel<<<1, 1>>>(N, d_x, i, d_Lx, d_inv_diag);
        hipDeviceSynchronize();

        blocks = (N - i + t - 1) / t;
        pre_comp_kernel<<<blocks, t>>>(d_L, d_Lx, d_x, i, N);
        hipDeviceSynchronize();
    }

    hipFree(d_inv_diag);
    hipFree(d_b_prod_inv_diag);
    hipFree(d_Lx);
    hipDeviceSynchronize();
}

void reset(double *L, double *x, double *b, double *d_L, double *d_x, double *d_b, int N) {
    hipMemcpy(d_L, L, N * N * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_x, x, N * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, N * sizeof(double), hipMemcpyHostToDevice);
    hipDeviceSynchronize();
}

void run_bm(int N, const char *preset) {
    double *L = (double*)malloc(sizeof(double) * N * N);
    double *x = (double*)malloc(sizeof(double) * N);
    double *b = (double*)malloc(sizeof(double) * N);

    double *d_L, *d_x, *d_b;
    hipMalloc((void **)&d_L, N * N * sizeof(double));
    hipMalloc((void **)&d_x, N * sizeof(double));
    hipMalloc((void **)&d_b, N * sizeof(double));

    if (VALIDATE) {
        reset(L, x, b, d_L, d_x, d_b, N);

        hipMemcpy(d_x, x, N * sizeof(double), hipMemcpyHostToDevice);
        hipDeviceSynchronize();
        kernel(d_L, d_x, d_b, N);

        hipMemcpy(x, d_x, N * sizeof(double), hipMemcpyDeviceToHost);
        hipDeviceSynchronize();

        hipError_t cudaErr = hipGetLastError();
        if (cudaErr != hipSuccess) {
            fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(cudaErr));
        }

        if (!is_correct(N, x, preset)) {
            printf("Validation failed for preset: %s \n", preset);
            exit(1);
        } else {
            printf("Validation passed for preset: %s \n", preset);
        }
    }

    dphpc_time3(
        reset(L, x, b, d_L, d_x, d_b, N),
        kernel(d_L, d_x, d_b, N),
        preset
    );

    hipFree(d_L);
    hipFree(d_x);
    hipFree(d_b);
    free(L);
    free(x);
    free(b);
}

int main() {
    run_bm(2000, "S");
    run_bm(5000, "M");
    run_bm(14000, "L");
    run_bm(16000, "paper");

    return 0;
}
