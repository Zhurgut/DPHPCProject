#include "hip/hip_runtime.h"

#include "../../timing/dphpc_timing.h"

#define get(A, ncols, r, c) A[(r)*(ncols)+(c)]

__global__ void kernel(int n, double *A, double *B, double *C) {
    int c = blockIdx.x * blockDim.x + threadIdx.x;
    int r = blockIdx.y * blockDim.y + threadIdx.y;
    if (r < n && c < n) {
        double s = 0.0;
        for (int i = 0; i < n; i++) {
            s += get(A, n, r, i) * get(B, n, i, c);
        }
        get(C, n, r, c) += s;
    }
}

void init_array(int n, double *A, double *B, double *out)
{
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++) {
            get(A, n, i, j) = (double) ((i+1)*(j+1)*3 % n);
            get(B, n, i, j) = (double) ((i+1)*(j+1)*7 % n);
            get(out, n, i, j) = 0.0;
        }
    }
}


void run_kernel(int n, double *A, double *B, double *C) {
    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((n-1) / 16 + 1, (n-1) / 16 + 1);
    kernel<<<numBlocks,threadsPerBlock>>>(n, A, B, C); 
    hipDeviceSynchronize();
}


void run_bm(int n, const char* preset) {

    double *A = (double*) malloc(n*n*sizeof(double));
    double *B = (double*) malloc(n*n*sizeof(double));
    double *C = (double*) malloc(n*n*sizeof(double));
    
    double *A_d, *B_d, *C_d;
    hipMalloc((void**) &A_d, n*n*sizeof(double));
    hipMalloc((void**) &B_d, n*n*sizeof(double));
    hipMalloc((void**) &C_d, n*n*sizeof(double));
    
    init_array(n, A, B, C);

    hipMemcpy((void*) A_d, (void*) A, n*n*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy((void*) B_d, (void*) B, n*n*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy((void*) C_d, (void*) C, n*n*sizeof(double), hipMemcpyHostToDevice);
    
    dphpc_time3(
        hipMemcpy(C_d, C, n*n*sizeof(double), hipMemcpyHostToDevice),
        run_kernel(n, A_d, B_d, C_d),
        preset
    );

    hipFree((void*) A_d);
    hipFree((void*) B_d);
    hipFree((void*) C_d);
    
    free((void*)A);
    free((void*)B);
    free((void*)C);
    
}

int is_valid() {

    int n = 50;

    double *A = (double*) malloc(n*n*sizeof(double));
    double *B = (double*) malloc(n*n*sizeof(double));
    double *C = (double*) malloc(n*n*sizeof(double));
    

    double *A_d, *B_d, *C_d;
    hipMalloc((void**) &A_d, n*n*sizeof(double));
    hipMalloc((void**) &B_d, n*n*sizeof(double));
    hipMalloc((void**) &C_d, n*n*sizeof(double));
    
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++) {
            get(A, n, i, j) = 1.0;
            get(B, n, i, j) = 1.0;
            get(C, n, i, j) = 0.0;
        }
    }

    hipMemcpy((void*) A_d, (void*) A, n*n*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy((void*) B_d, (void*) B, n*n*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy((void*) C_d, (void*) C, n*n*sizeof(double), hipMemcpyHostToDevice);
    


    run_kernel(n, A_d, B_d, C_d);


    hipMemcpy((void*) C, (void*) C_d, n*n*sizeof(double), hipMemcpyDeviceToHost);

    hipFree((void*) A_d);
    hipFree((void*) B_d);
    hipFree((void*) C_d);

    free((void*)A);
    free((void*)B);



    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++) {
            if (get(C, n, i, j) != n) {
                free((void*)C);
                return 0;
            }
        }
    }
    free((void*)C);
    return 1;
}


#include "_main.h"


