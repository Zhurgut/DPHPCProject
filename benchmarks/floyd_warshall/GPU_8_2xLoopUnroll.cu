
#include <hip/hip_runtime.h>
/* DOES NOT RESOLVE RACE CONDITION - BUGGY

#include "utils.h"

#define ASSERT 1


__global__ void kernel_floyd_warshall(int n, int *graph) {

  int tmp, tmp1;
  int j = blockIdx.x * blockDim.x + threadIdx.x;
  int i = blockIdx.y * blockDim.y + threadIdx.y;

  if (i < n && j < n) {
    for (int k = 0; k < n-1; k+=2){
      tmp = graph[i * n + k] + graph[k * n + j];
      tmp1 = graph[i * n + k + 1] + graph[(k + 1) * n + j];
      if (tmp > tmp1) tmp = tmp1;

      if (tmp < graph[i * n + j]) {
        graph[i * n + j] = tmp;
      }
    }
    tmp = graph[i * n + (n - 1)] + graph[(n - 1) * n + j];
    if (tmp < graph[i * n + j]) {
      graph[i * n + j] = tmp;
    }
  }
}


void run_floyd_warshall_gpu(int n, int *graph) {
  
  dim3 threadsPerBlock(16, 16);
  dim3 numBlocks(n / 16 + 1, n / 16 + 1);
  kernel_floyd_warshall<<<numBlocks,threadsPerBlock>>>(n, graph);
  cudaDeviceSynchronize();
}


int main(int argc, char** argv) {
  
  run_bm(N_S, "S", run_floyd_warshall_gpu, ASSERT);
  run_bm(N_M, "M", run_floyd_warshall_gpu, ASSERT);
  run_bm(N_L, "L", run_floyd_warshall_gpu, ASSERT);
  run_bm(N_PAPER, "paper", run_floyd_warshall_gpu, ASSERT);
  
  return 0;
}
*/