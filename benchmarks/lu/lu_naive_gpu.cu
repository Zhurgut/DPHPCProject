#include "hip/hip_runtime.h"
#include <stdio.h>
#include <unistd.h>
#include <math.h>
#include <stdlib.h>

#include "../../timing/dphpc_timing.h"



__global__ void lu_kernal(int N, double* A){

 int i = blockIdx.x * blockDim.x + threadIdx.x;
 int j, k;

    if (i < N) {
        for (j = 0; j < i; j++) {
            for (k = 0; k < j; k++) {
                A[i * N + j] = A[i * N + j] - (A[i * N + k] * A[k * N + j]);
            }
            A[i * N + j] = A[i * N + j] / A[j * N + j];
        }
        for (j = i; j < N; j++) {
            for (k = 0; k < i; k++) {
                A[i * N + j] = A[i * N + j] - (A[i * N + k] * A[k * N + j]);
            }
        }
    }
}


void init_array(int N, double* A) {

  double* B = (double*)malloc(N * N * sizeof(double));

  // create lower triangle of matrix 
  for (int i = 0; i < N; i++) {
    // initialize the lower triangle 
    for (int j = 0; j <= i; j++) {
      A[i * N + j] = (double)(-j % N) / N + 1;
    }
    // set upper triangle to zero 
    for (int j = i + 1; j < N; j++) {
      A[i * N + j] = 0;
    }
    // set elements on the diagonal to 1 
    A[i * N + i] = 1;
  }

  // multiply A by A^T and save the result in B
  // result is a symmetric matrix 
  for (int t = 0; t < N; ++t)
    for (int r = 0; r < N; ++r)
      for (int s = 0; s < N; ++s)
        B[r * N + s] = B[r * N + s] + (A[r * N + t] * A[s * N + t]);

  // Copy the result back to A
  for (int r = 0; r < N; ++r)
    for (int s = 0; s < N; ++s)
      A[r * N + s] = B[r * N + s];

  // Free the dynamically allocated memory for A and B
  free(B);
}




void run_lu_kernel(int N, double* A) {
    int block = 16;
    int grid = (N+block -1)/block;
    lu_kernal<<<grid,block>>>(N, A); 
    hipDeviceSynchronize();
}

void run_bm(int N, const char* preset) {

    double* A = (double *)malloc(N*N*sizeof(double));
    double *A_d;

    hipMalloc((void**) &A_d, N*N*sizeof(double));
   
    init_array(N, A);

    
    hipMemcpy((void*) A_d, (void*) A, N*N*sizeof(double), hipMemcpyHostToDevice);
    
    dphpc_time3(
       hipMemcpy((void*) A_d, (void*) A, N*N*sizeof(double), hipMemcpyHostToDevice),
       run_lu_kernel(N, A_d),
        preset
    );

    hipFree(A_d);
  
    free(A);

}

int main(int argc, char** argv) {

    run_bm(60, "S"); 
    run_bm(220, "M"); 
    run_bm(700, "L"); 
    run_bm(2000, "paper"); 

  return 0;
}



