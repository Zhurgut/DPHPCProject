#include "hip/hip_runtime.h"

#include "../../timing/dphpc_timing.h"

__global__ void gpufib(int n, int* out) {
    if (n <= 2) {
        out[0] = 1;
    } else {
        int sn = 1;
        int sn_1 = 1;
        for (int i = 3; i <= n; i++) {
            int tmp = sn_1;
            sn_1 = sn;
            sn = sn + tmp;
        }
        out[0] = sn;
    }
}

void run_kernel(int n, int* out) {
    gpufib<<<1, 1>>>(n, out); // measure performance of single gpu core
    hipDeviceSynchronize(); // wait until kernel is done, otherwise just measuring how long it takes to launch the kernel
}

int main() {
    int* out;
    hipMalloc((void**) &out, sizeof(int));


    dphpc_time(
        run_kernel(38, out);
    );
    
    // printf("%s\n", hipGetErrorString(hipGetLastError()));

    
    // hipMemcpy(&result, out, sizeof(int), hipMemcpyDeviceToHost);
    // printf("result after: %d\n", result);

    dphpc_time3(
        ,
        run_kernel(1000, out),
        "S"
    );

    dphpc_time3(
        ,
        run_kernel(10000, out),
        "M"
    );

    dphpc_time3(
        ,
        run_kernel(100000, out),
        "L"
    );

    dphpc_time3(
        ,
        run_kernel(100500, out),
        "paper"
    );

    hipFree(out);
}
