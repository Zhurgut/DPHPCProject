#include "hip/hip_runtime.h"
#include "utils.h"

#define ASSERT 1
#define THREADS 16


__global__ void kernel_doitgen(int nr, int nq, int np,
      double *A,
      double *C4,
      double *sum) {

    // Very bad performance - optimization replace writing to sum with temporary variable
    int r = blockIdx.x * blockDim.x + threadIdx.x;
    int q = blockIdx.y * blockDim.y + threadIdx.y;

    if (r < nr && q < nq) {
        for(int p = 0; p < np; p++) {
            for (int s = 0; s < np; s++) {
                sum[r * nq * np + q * np + s] = sum[r * nq * np + q * np + s] + A[r * nq * np + q * np + p] * C4[p * np + s];
            }
        }
        for (int p = 0; p < np; p++) {
            A[r * nq * np + q * np + p] = sum[r * nq * np + q * np + p];
        }
    }
}


void run_doitgen_gpu(int nr, int nq, int np,
      double *A,
      double *C4,
      double *sum) {

    dim3 threadsPerBlock(THREADS, THREADS);
    dim3 numBlocks(CEIL_DIV(nr, THREADS), CEIL_DIV(nq, THREADS));
    kernel_doitgen<<<numBlocks,threadsPerBlock>>>(nr, nq, np, A, C4, sum);
    hipDeviceSynchronize();
}


int main(int argc, char** argv) {

    int nr = NR_S; int nq = NQ_S; int np = NP_S;
    run_bm(nr, nq, np, "S", run_doitgen_gpu, ASSERT);
    
    nr = NR_M; nq = NQ_M; np = NP_M;
    run_bm(nr, nq, np, "M", run_doitgen_gpu, ASSERT);

    nr = NR_L; nq = NQ_L; np = NP_L;
    run_bm(nr, nq, np, "L", run_doitgen_gpu, ASSERT);

    nr = NR_PAPER; nq = NQ_PAPER; np = NP_PAPER;
    run_bm(nr, nq, np, "paper", run_doitgen_gpu, ASSERT);

    return 0;
}
